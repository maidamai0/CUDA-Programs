#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// example 5.1 rotate1  This version uses bilinear interpolation on GPU
// 
// RTX 2070
// C:\bin\rotate1.exe data\ives512.raw data\test.raw 512 512 1.0 10000
// file data\ives512.raw read
// file data\test.raw written
// rotate1 iterations 10000 time 83.691 ms
// 
// RTX 3080
// C:\bin\rotate1.exe data\ives512.raw data\test.raw 512 512 1.0 10000
// file data\ives512.raw read
// file data\test.raw written
// rotate1 iterations 10000 time 47.140 ms 

#include "cx.h"
#include "hip/hip_vector_types.h"  // for lerp
#include "cxtimers.h"
#include "cxbinio.h"

// this version assumes (x,y) are coodinates of pixel CENTRE
template <typename T> __host__ __device__ T bilinear(cr_Ptr<T> a,float x,float y,int nx,int ny)
{
	auto idx = [&nx](int y,int x){ return y*nx+x; };
	if(x < -1.0f ||  x >= nx || y < -1.0f || y >= ny) return (T)0;

	float x1 = floorf(x-0.5f); // (x,y) is the
	float y1 = floorf(y-0.5f); // pixel CENTRE
	float ax = x - x1 - 0.5f;  // gap between pixel left sides
	float ay = y - y1 - 0.5f;  // gap between pixel bottoms

	int kx1 = max(0,(int)x1); int kx2 = min(nx-1,kx1+1);
	int ky1 = max(0,(int)y1); int ky2 = min(ny-1,ky1+1);

	float ly1 = lerp(a[idx(ky1,kx1)],a[idx(ky1,kx2)],ax);  // x interp at y1
	float ly2 = lerp(a[idx(ky2,kx1)],a[idx(ky2,kx2)],ax);  // x interp at y2
	return (T)lerp(ly1,ly2,ay);     // y interp of the x interpolated values              
}

// this version assumes (x,y) are coodinates of pixel lower left hand CORNER
template <typename T> __host__ __device__ T bilinear_corner(cr_Ptr<T> a,float x,float y,int nx,int ny)
{
	auto idx = [&nx](int y,int x){ return y*nx+x; };
	if(x < -1.0f ||  x >= nx || y < -1.0f || y >= ny) return (T)0;

	float x1 = floorf(x); // (x,y) is lower
	float y1 = floorf(y); // left hand CORNER
	float ax = x - x1;   // gap between left sides
	float ay = y - y1;   // gap between bottoms

	int kx1 = max(0,(int)x1); int kx2 = min(nx-1,kx1+1);
	int ky1 = max(0,(int)y1); int ky2 = min(ny-1,ky1+1);

	float ly1 = lerp(a[idx(ky1,kx1)],a[idx(ky1,kx2)],ax);  // x interp at y1
	float ly2 = lerp(a[idx(ky2,kx1)],a[idx(ky2,kx2)],ax);  // x interp at y2
	return (T)lerp(ly1,ly2,ay);     // y interp of the x interpolated values              
}


template <typename T> __global__ void rotate1(r_Ptr<T> b,cr_Ptr<T> a,float angle,int nx,int ny)

{
	cint x = blockIdx.x*blockDim.x + threadIdx.x;
	cint y = blockIdx.y*blockDim.y + threadIdx.y;
	if(x >= nx || y >= ny) return; // Check if within image bounds
	auto idx = [&nx](int y,int x){ return y*nx+x; };

	float xt = x - nx/2.0f;  // translate to make the centre of the 
	float yt = y - ny/2.0f;  // image the centre of rotation
	float xr =  xt*cosf(angle)+ yt*sinf(angle) + nx/2.0f;
	float yr = -xt*sinf(angle)+ yt*cosf(angle) + ny/2.0f;

	// choose one of these interpoalaton functions
	b[idx(y,x)] = bilinear(a,xr,yr,nx,ny);
	//b[idx(y,x)] = bilinear_corner(a,xr,yr,nx,ny);
}

int main(int argc,char *argv[])
{
	if(argc <3){
		printf("usage rotate1 <infile> <outfile> nx|512 ny|512 angle|30 iterations|1");
		return 0;
	}
	int nx      = (argc > 3) ? atoi(argv[3]) : 512;
	int ny      = (argc > 4) ? atoi(argv[4]) : nx;
	float angle = (argc > 5) ? atoi(argv[5]) : 30.0f;
	int iter    = (argc > 6) ? atoi(argv[6]) : 1;
	angle *= cx::pi<float>/180.0f;  // to radians
	int size = nx*ny;

	thrustHvec<uchar> a(size);
	thrustHvec<uchar> b(size);
	thrustDvec<uchar> dev_a(size);
	thrustDvec<uchar> dev_b(size);

	if(cx::read_raw(argv[1],a.data(),size)) return 1;
	dev_a = a;  // copy to device

	dim3 threads ={16,16,1};
	dim3 blocks ={(uint)(nx+15)/16,(uint)(ny+15)/16,1};
	cx::timer tim;
	for(int k=0;k<iter;k++){
		rotate1<uchar><<<blocks,threads>>>(dev_b.data().get(),dev_a.data().get(),angle,nx,ny);
	}

        cx::cudaOK(hipGetLastError());
        cx::cudaOK(hipDeviceSynchronize());
        double t1 = tim.lap_ms();

	b = dev_b; // get results
	cx::write_raw(argv[2],b.data(),size);
	printf("rotate1 iterations %d time %.3f ms\n",iter,t1);

	std::atexit([]{hipDeviceReset();});  // thrust safe reset
	return 0;
}