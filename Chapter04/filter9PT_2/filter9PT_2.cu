#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// example 4.9 filter9PT_2
// 
// RTX 2070
// C:\bin\filter9PT_2.exe data\ives512.raw data\test.raw 512 512 16 16 10000 1.0 1.0 1.0   1.0 1.0 1.0   1.0 1.0 1.0
// file data\ives512.raw read
// filter9PT_2 iterations 10000 time 103.863 ms
// file data\test.raw written
// 
// RTX 3080
// C:\bin\filter9PT_2.exe data\ives512.raw data\test.raw 512 512 16 16 10000 1.0 1.0 1.0   1.0 1.0 1.0   1.0 1.0 1.0
// file data\ives512.raw read
// filter9PT_2 iterations 10000 time 47.575 ms
// file data\test.raw written

#include "cx.h"
#include "cxbinio.h"
#include "cxtimers.h"

// data explicilty in constnat memory must be declared at file scope
// arrays sizes must be known at compile time.
__constant__ float fc[9];  

// filter constants stored in device global memory
__global__ void filter9PT_2(cr_Ptr<uchar> a,r_Ptr<uchar> b,int nx,int ny)
{
	auto idx = [&nx](int y,int x){ return y*nx+x; };

	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	if(x<0 || y <0 || x >= nx || y >= ny)return;
	int xl = max(0,x-1); int yl = max(0,y-1);
	int xh = min(nx-1,x+1); int yh = min(ny-1,y+1);

	float v = fc[0]*a[idx(yl,xl)] + fc[1]*a[idx(yl,x)] + fc[2]*a[idx(yl,xh)] +
		      fc[3]*a[idx(y,xl)]  + fc[4]*a[idx(y,x)]  + fc[5]*a[idx(y,xh)]  +
		      fc[6]*a[idx(yh,xl)] + fc[7]*a[idx(yh,x)] + fc[8]*a[idx(yh,xh)];

	uint f = (uint)(v+0.5f);
	b[idx(y,x)] = (uchar)min(255,max(0,f)); // b in [0,255]
}

int main(int argc,char *argv[])
{
	if(argc < 2){
		printf("usage filter9PT_2 <in file> <outfile> nx|512 ny|512 tx|16 ty|16 iter|10000 c[0] ... c[9]\n");
	}
	thrustHvec<float> c(9);

	uint nx      = (argc > 3) ? atoi(argv[3]) : 512;
	uint ny      = (argc > 4) ? atoi(argv[4]) : 512;
	uint threadx = (argc > 5) ? atoi(argv[5]) : 16;
	uint thready = (argc > 6) ? atoi(argv[6]) : 16;
	uint iter    = (argc > 7) ? atoi(argv[7]) : 10000; // this for timing
	for(int k=0;k<9;k++) c[k] = (argc > 8+k) ? atof(argv[8+k]) : 1.0;
	uint size = nx*ny;

        cx::cudaOK(hipSetDevice(0));  // Choose which GPU to run on

        thrustHvec<uchar> a(size);
	thrustHvec<uchar> b(size);
	thrustDvec<uchar> dev_a(size);
	thrustDvec<uchar> dev_b(size);

	// normalise filter coefficients
	float csum = 0.0f;
	for(int k=0;k<9;k++) csum += c[k];
	if(fabs(csum) > 0.001) for(int k=0;k<9;k++) c[k] /= csum;

	if(cx::read_raw(argv[1],a.data(),size)) return 1;
	hipMemcpyToSymbol(HIP_SYMBOL(fc), c.data(),9*sizeof(float));  // replaces dev_c = c
	dev_a = a;

	dim3 threads ={threadx,thready,1};
	dim3 blocks ={(nx+threads.x-1)/threads.x,(ny+threads.y-1)/threads.y,1};

	cx::timer tim;
	for(uint k=0;k<iter;k++){  //filterered version of a placed in b on each pass
		filter9PT_2<<<blocks,threads>>>(dev_a.data().get(),dev_b.data().get(),nx,ny);
	}

	hipDeviceSynchronize();
	double t1 = tim.lap_ms();
	printf("filter9PT_2 iterations %d time %.3f ms\n",iter,t1);

	// save result in b
	b = dev_b;
	cx::write_raw(argv[2],b.data(),size);

	std::atexit([]{hipDeviceReset();});  // thrust safe reset
	return 0;
}
