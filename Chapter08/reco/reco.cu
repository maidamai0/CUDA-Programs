#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// reco program includes examples 8.7 and 8.8.
// 
// RTX 2070
// using blocks = 180000, threads = 400 valcut 0.0 dzcut 63
// file petSMtab.raw read
// sm_size = 12829371, lor_size 167232000 vol_size 2560000
// file petSM.raw read
// file derenzo_full.raw read
// normalization done for 100 rings and 64 slices
// file norm_new.raw written
// file smnorm.raw written
// Host normalize call 218.755 ms
// total activity 33232349840, activity density 264455
// iteration   1 chi 6.187530e+16 times fwd 1738.016 bwd 1713.315 rsc 0.166 chi 0.339 all 3451.838 ms
// iteration   2 chi 5.228258e+16 times fwd 3434.485 bwd 3438.568 rsc 0.325 chi 0.656 all 6874.229 ms
// iteration   3 chi 4.512538e+16 times fwd 5133.819 bwd 5166.416 rsc 0.485 chi 0.972 all 10302.026 ms
// iteration   4 chi 3.954603e+16 times fwd 6832.397 bwd 6896.475 rsc 0.651 chi 1.286 all 13731.291 ms
// iteration   5 chi 3.499340e+16 times fwd 8537.485 bwd 8631.004 rsc 0.809 chi 1.601 all 17171.546 ms
// iteration   6 chi 3.122855e+16 times fwd 10241.888 bwd 10367.569 rsc 0.980 chi 1.876 all 20613.123 ms
// iteration   7 chi 2.809372e+16 times fwd 11947.100 bwd 12106.307 rsc 1.142 chi 2.222 all 24057.757 ms
// iteration   8 chi 2.546026e+16 times fwd 13656.071 bwd 13849.784 rsc 1.301 chi 2.537 all 27510.837 ms
// iteration   9 chi 2.322437e+16 times fwd 15367.685 bwd 15595.321 rsc 1.452 chi 2.854 all 30968.618 ms
// iteration  10 chi 2.130530e+16 times fwd 17081.295 bwd 17342.082 rsc 1.613 chi 3.171 all 34429.630 ms
// file reco2022cav_mlem010.raw written
// All time 34435.392 ms
// 
// RTX 3080
// C:\bin\reco.exe derenzo_full.raw recotest petSM.raw petSMtab.raw 10
// using blocks = 180000, threads = 400 valcut 0.0 dzcut 63
// file petSMtab.raw read
// sm_size = 12829371, lor_size 167232000 vol_size 2560000
// file petSM.raw read
// file derenzo_full.raw read
// bad open on gold.raw for read
// normalization done for 100 rings and 64 slices
// file norm_new.raw written
// file smnorm.raw written
// Host normalize call 194.542 ms
// total activity 33232349840, activity density 264455
// iteration   1 chi 6.187530e+16 times fwd 1042.826 bwd 955.639 rsc 0.057 chi 0.152 all 1998.676 ms
// iteration   2 chi 5.228258e+16 times fwd 2076.026 bwd 1920.139 rsc 0.113 chi 0.281 all 3996.642 ms
// iteration   3 chi 4.512538e+16 times fwd 3111.229 bwd 2886.575 rsc 0.172 chi 0.412 all 5998.537 ms
// iteration   4 chi 3.954603e+16 times fwd 4146.787 bwd 3854.938 rsc 0.228 chi 0.541 all 8002.729 ms
// iteration   5 chi 3.499340e+16 times fwd 5182.085 bwd 4824.915 rsc 0.282 chi 0.672 all 10008.276 ms
// iteration   6 chi 3.122855e+16 times fwd 6217.956 bwd 5794.162 rsc 0.335 chi 0.806 all 12013.653 ms
// iteration   7 chi 2.809372e+16 times fwd 7252.934 bwd 6765.277 rsc 0.392 chi 0.939 all 14020.012 ms
// iteration   8 chi 2.546026e+16 times fwd 8288.410 bwd 7736.630 rsc 0.455 chi 1.072 all 16027.124 ms
// iteration   9 chi 2.322437e+16 times fwd 9325.056 bwd 8708.379 rsc 0.515 chi 1.211 all 18035.805 ms
// iteration  10 chi 2.130530e+16 times fwd 10361.244 bwd 9680.231 rsc 0.599 chi 1.356 all 20044.162 ms
// file recotest.raw_mlem010.raw written
// All time 20048.690 ms

#include "cx.h"
#include "cxtimers.h"
#include "cxbinio.h"
#include "scanner.h"


__host__ __device__ int c2_to_dc2(cint c1,cint c2) {
	return cyc_sub(c2,c1)-cryDiffMin;
}

//NB this can be called with either z1 or (z2-z1) as argument
//   steps in the other variable will then be adjacent in memory
//   Using (z2-z1) are argument turns out to be a bit faster.
__host__ __device__ int  zdz_slice(int z)
{
	return detZdZNum - (zNum-z)*(zNum-z+1)/2;
}

// Host version of example 8.7
// assumes threads = cryNum i.e. 400 so that one thread blocks processes all the phis for single sm value
int forward_project_host(smPart *sm,uint smstart,uint smend,uint *meas,float* vol,int ring,float* FP,int dzcut,float valcut)
{
	uint smpos = smstart;
	//uint count = 0;
	while(smpos < smend) {
		for(int phi=0;phi<cryNum;phi++){  // for loop over threads
			smLor tl = key2lor(sm[smpos].key);
			tl.c1 = cyc_add(tl.c1,phi);     // rotate by phi		
			tl.c2 = cyc_add(tl.c2,phi);     // rotate by phi
			if(tl.zsm1==0 && tl.zsm2==0 && tl.c2 <= tl.c1) continue;  // skip unused case
			int dc = c2_to_dc2(tl.c1,tl.c2);         // sm has actual c2 not delta	
			int dz = tl.zsm1+tl.zsm2;
			float val= sm[smpos].val;
			if(dz > dzcut || val <valcut) break;
			uint lor_index = zdz_slice(dz)*cryCdCNum + dc*cryNum + tl.c1;
			uint vol_index = (ring*zNum + tl.zsm1)*cryNum + phi;      // z+z1 here as voxel index
			for(int zs1 = 0; zs1 < zNum-dz; zs1++) {  // zs1 is sliding posn of lh end of lor 
				if(meas[lor_index]>0){
					float element = vol[vol_index] * val;
					FP[lor_index] += element;
				}
				lor_index += cryCdCNum;  // for zs1++
				vol_index += cryNum;     // for zs1++
			}
		}   // end phi loop
		smpos++;  // one host thread
	}
	return 0;
}

// Host version of example 8.8
int backward_project_host(smPart* sm,uint smstart,uint smend,uint *meas,int ring,float* FP,float* BP,int dzcut,float valcut)
{
	uint smpos = smstart;
	while(smpos < smend) {
		for(int phi=0;phi<cryNum;phi++){  // for loop over threads
			smLor tl = key2lor(sm[smpos].key);
			tl.c1 = cyc_add(tl.c1,phi);     // rotate by phi		
			tl.c2 = cyc_add(tl.c2,phi);     // rotate by phi
			if(tl.zsm1==0 && tl.zsm2==0 && tl.c2 <= tl.c1) continue; // skip unused case
			int dc = c2_to_dc2(tl.c1,tl.c2);         // sm has actual c2 not delta		
			int dz = tl.zsm1+tl.zsm2;  // net delta z
			float val= sm[smpos].val;
			if(dz > dzcut || val < valcut) break;
			uint lor_index = zdz_slice(dz)*cryCdCNum + dc*cryNum + tl.c1;  // new july 6 
			uint vol_index = (ring*zNum + tl.zsm1)*cryNum + phi;       // z1+zs1 here as voxel index
			for(int zs1 = 0; zs1 < zNum-dz; zs1++) {  // zs1 is sliding posn of lh end of lor 
				if(meas[lor_index]>0){
					//if(FP[lor_index == 0.0f]) printf("zero FP  found index %u\n",lor_index);
					float element = val * meas[lor_index] / FP[lor_index];  // val added 27/06/19!!
					BP[vol_index] += element;
				}
				lor_index += cryCdCNum;  // for zs1++
				vol_index += cryNum;     // for zs1++
			}
		}
		smpos++;
	}
	return 0;
}

int rescale_host(r_Ptr<float> vol,cr_Ptr<float> BP,cr_Ptr<float> norm)
{
	for(int id=0;id<zNum*radNum;id++){
		float scale = 1.0f/norm[id];
		for(int phi=0;phi<cryNum;phi++) vol[id*cryNum+phi] *= BP[id*cryNum+phi]*scale;
	}
	return 0;

}

// example 8.7 (GPU version)
// Uses thread linear addressing for flexible thread bock sizes.
__global__  void forward_project(cr_Ptr<smPart> sm,uint smstart,uint smend,cr_Ptr<float> vol,int ring,r_Ptr<float> FP,int dzcut,float valcut)
{
	// This version uses thread linar addressing to allow tuning experiments
	uint id = blockIdx.x*blockDim.x+threadIdx.x;
	uint tstride = gridDim.x*blockDim.x;
	int nphi = (smend-smstart)*cryNum;
	while(id < nphi) {
		int phi =id%cryNum;                // these two lines added for
		int smpos = smstart+ id/cryNum;    // thread linear addressing
		smLor tl = key2lor(sm[smpos].key);
		tl.c1 = cyc_add(tl.c1,phi);        // rotate by phi		
		tl.c2 = cyc_add(tl.c2,phi);        // rotate by phi
		int dc = c2_to_dc2(tl.c1,tl.c2);   // sm has actual c2 not delta	
		int dz = tl.zsm1+tl.zsm2;
	
		float val= sm[smpos].val;  // system matrix value
		if(dz > dzcut || val <valcut) { smpos += gridDim.x; continue; }

		uint lor_index = zdz_slice(dz)*cryCdCNum + dc*cryNum + tl.c1;
		uint vol_index = (ring*zNum + tl.zsm1)*cryNum + phi;      // z+z1 here as voxel index
		for(int zs1 = 0; zs1 < zNum-dz; zs1++) {  // zs1 is sliding posn of lh end of lor
			float element = vol[vol_index] * val;
			atomicAdd(&FP[lor_index],element);
			lor_index += cryCdCNum;  // for zs1++
			vol_index += cryNum;     // for zs1++
		}
		id += tstride;  // replaces smpos+=gridDim.x for thread linear addressing case
	}
}

// Example 8.8 (GPU version)
// Uses thread linear addressing for flexible thread bock sizes.
__global__  void backward_project(cr_Ptr<smPart> sm,uint smstart,uint smend,cr_Ptr<uint> meas,int ring,cr_Ptr<float> FP,r_Ptr<float> BP,int dzcut,float valcut)
{
	// This version uses thread linar addressing to allow tuning experiments
	uint id = blockIdx.x*blockDim.x+threadIdx.x;
	uint tstride = gridDim.x*blockDim.x;
	int nphi = (smend-smstart)*cryNum;
	while(id < nphi) {
		int phi =id%cryNum;                 // these two lines added for
		int smpos = smstart+id/cryNum;      // thread linear addressing
		smLor tl = key2lor(sm[smpos].key);
		tl.c1 = cyc_add(tl.c1,phi);         // rotate by phi		
		tl.c2 = cyc_add(tl.c2,phi);         // rotate by phi
		int dc = c2_to_dc2(tl.c1,tl.c2);    // sm has actual c2 not delta		
		int dz = tl.zsm1+tl.zsm2;  
	
		float val= sm[smpos].val;   // system matrix value
		if(dz > dzcut || val < valcut) { smpos += gridDim.x; continue; }

		uint lor_index = zdz_slice(dz)*cryCdCNum + dc*cryNum + tl.c1;  // new july 6 
		uint vol_index = (ring*zNum + tl.zsm1)*cryNum + phi;       // z1+zs1 here as voxel index
		for(int zs1 = 0; zs1 < zNum-dz; zs1++) {  // zs1 is sliding posn of lh end of lor 	
			float FPdiv = max(1.0f,FP[lor_index]);
			float element = val * meas[lor_index] / FPdiv;
			atomicAdd(&BP[vol_index],element);
			lor_index += cryCdCNum;  // for zs1++
			vol_index += cryNum;     // for zs1++
		}
		id += tstride;  // replaces smpos+=gridDim.x for thread linear addressing case
	}
}

__global__ void rescale(r_Ptr<float> vol,cr_Ptr<float> BP,cr_Ptr<float> norm)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	while(id < zNum*radNum*cryNum){

		vol[id] *= BP[id] / norm[id/cryNum];
		id += blockDim.x*gridDim.x;
	}
}

__global__ void calc_chisd(r_Ptr<float> vol,cr_Ptr<float> gold,r_Ptr<float> chisd)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	while(id < zNum*radNum*cryNum){
		chisd[id] = (vol[id]-gold[id])*(vol[id]-gold[id]);
		id += blockDim.x*gridDim.x;
	}
}

template <typename T> __global__ void clear_vector(r_Ptr<float> a,uint len)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	while(id < len){
		a[id] = (T)0;
		id += blockDim.x*gridDim.x;
	}
}

//int normalise_sm(thrustHvec<smPart> &sm,thrustHvec<float> &norm,thrustHvec<uint> &smhits,thrustHvec<uint> &smstart, int dzcut, float valcut)
int normalise_sm(thrustHvec<smPart> &sm,thrustHvec<float> &norm,thrustHvec<smTab> &systab,int dzcut,float valcut)
{
	// NB all dZ and dC cuts assumed to have already been made in readspot
	uint norm_size = radNum*zNum;
	// normalise allowing for voxel volume Router^2 - Rinner^2
	for(int r = 0;r<radNum;r++){
		//uint sm_start = smstart[r];
		//uint smnum = smhits[r+2];
		//for(uint k=sm_start;k<sm_start+smnum;k++){
		for(uint k=systab[r].start;k<systab[r].end;k++){
			smLor tl = key2lor(sm[k].key);
			float val = sm[k].val;
			int dz = tl.zsm2+tl.zsm1;  // changed 9/7/19
			if(dz > dzcut || val < valcut) continue;
			for(int z = tl.zsm1; z < zNum - tl.zsm2; z++)  norm[r*zNum+z] += val;	  //vertex posn here
		}
	}
	printf("normalization done for %d rings and %d slices\n",radNum,zNum);
	for(uint i=0;i<norm_size;i++) norm[i] /= sysMatScale;  // assume 10^10 generations per voxel
	cx::write_raw("norm_new.raw",norm.data(),norm_size);
	//cx::write_raw("norm_recip.raw",norm.data(),norm_size);
	return 0;
}

int list_sm(thrustHvec<smPart> &sm,thrustHvec<uint> &smhits,thrustHvec<uint> &smstart)
{
	printf("list sm called\n");
	for(int r=0;r<radNum;r++){
		printf("list sm called r=%d\n",r);
		char name[256];
		sprintf(name,"smlist_r%3.3d.txt",r);
		FILE * flog = fopen(name,"w");
		uint sm_start = smstart[r];
		uint smnum = smhits[r+2];
		for(uint k=sm_start;k<sm_start+smnum;k++){
			smLor tl = key2lor(sm[k].key);
			float val = sm[k].val;
			fprintf(flog,"smpos %6u lor (%2d %3d)-(%2d %3d) val %.0f\n",k,tl.zsm1,tl.c1,tl.zsm2,tl.c2,val);
		}
		fclose(flog);
	}
	return 0;
}

// save direclty are Cartesian 200x200x64 image
int pol_save(const char* name, float* vol)
{
	struct cp_grid_map {
		float b[voxBox][voxBox];
		int x; // carteisian origin
		int y;
		int phi;  // polar voxel
		int r;
	};

	//int pol_size =  cryNum*zNum*radNum;  // NB order [ring, z, phi]
	int cart_size = voxNum*voxNum*zNum;  //          [2*z,    y,   x]
	int map_size =  cryNum*radNum;       //          [ring, phi]

	std::vector<float>       cart(cart_size);
	std::vector<cp_grid_map>  map(map_size);
	if(cx::read_raw("pol2cart.tab",map.data(),map_size,0)){printf("bad read on pol2cart.tab\n"); return 1;}
	for(int r=0;r<radNum;r++) for(int z=0;z<zNum;z++) for(int p=0;p<cryNum;p++){
		float val = vol[(r*zNum+z)*cryNum+p];

		float vol_fraction =  1.0f;  //2*r+1;
		int index = r*cryNum+p;
		if(val > 0.0f){
			int x0 = map[index].x;
			int y0 = map[index].y;
			for(int i=0;i<voxBox;i++) {
				int y = y0+i;
				if(y>=0 && y<voxNum) for(int j= 0;j<voxBox;j++){
					int x = x0+j;
					if(x>=0 && x <voxNum && map[index].b[i][j]>0.0f) cart[(z*voxNum+y)*voxNum+x] += vol_fraction*val*map[index].b[i][j];
				}
			}
		}
	}
	cx::write_raw(name,cart.data(),cart_size);
	return 0;
}

int main(int argc,char *argv[])
{
	if(argc < 2){
		printf("usage reco <pet file (phantom)> <result file> <sm file> <sm tab file> <iterations> [ dzcut|63] valcut|0 usehost|0] blocks|5000 threads|400 rmin|0 rmax|100  snapsave|50\n");
		return 0;
	}

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	int blscale = 1024;
	int thscale = 256;
	int niter    = (argc > 5) ? atoi(argv[5]) : 10;
	int dzcut    = (argc > 6) ? atoi(argv[6]) : 63;
	float valcut = (argc > 7) ? atof(argv[7]) : 0.0;
	int usehost  = (argc > 8) ? atoi(argv[8]) : 0; 
	if(usehost != 0) printf("WARNING HOST code being used for this run\n");
	
	// set up system matix
	char name[256];

	int blocks  = (argc> 9) ? atoi(argv[9])*36 : 5000*36;  // 36 =  number of sm units on gpu
	int threads = (argc > 10) ? atoi(argv[10]) : cryNum;
	printf("using blocks = %d, threads = %d valcut %.1f dzcut %d\n",blocks,threads,valcut,dzcut);

	int rmin  = (argc > 11) ? atoi(argv[11]) : 0;
	int rmax = (argc > 12) ? atoi(argv[12]) : 100;
	int snapsave = (argc > 13) ? atoi(argv[13]) : 50;  // extra saves

	// this for sysmat_tab
	thrustHvec<smTab> systab(radNum);  // start and end indicies for individual rings within sysmat file
	if(cx::read_raw(argv[4],systab.data(),radNum)){ printf("bad read %s\n",argv[4]); return 1; }

	uint sm_size = systab[radNum-1].end;
	uint lor_size = cryCdCNum*detZdZNum;
	uint vol_size = cryNum*radNum*zNum;
	uint norm_size = radNum*zNum;
	uint zphi_size = cryNum*zNum;
	printf("sm_size = %u, lor_size %u vol_size %u\n",sm_size,lor_size,vol_size);

	thrustHvec<smPart>      sm(sm_size);  // the system matrix (sysmat) file generated by fullsim + readspot
	thrustDvec<smPart>  dev_sm(sm_size);
	if(cx::read_raw(argv[3],sm.data(),sm_size)) { printf("bad read on sysmat file %s\n",argv[3]); return 1; }
	dev_sm = sm;

	thrustHvec<uint>      meas(lor_size);  // the PET measured lor file, either real mesurements 
	thrustDvec<uint>  dev_meas(lor_size);  // or simulated phantom data
	if(cx::read_raw(argv[1],meas.data(),lor_size)) { printf("bad read on PET measurment file %s\n",argv[1]); return 1; }
	dev_meas = meas;

	thrustHvec<float>     FP(lor_size); // working space for forward projection (voxels => lors)
	thrustDvec<float> dev_FP(lor_size);

	thrustHvec<float>     BP(vol_size); // working space for backward projection  (lors => voxels)
	thrustDvec<float> dev_BP(vol_size);

	thrustHvec<float>     vol(vol_size); // The PET voxels to be calculated
	thrustDvec<float> dev_vol(vol_size);

	// here we use the known generated phantom from fullsim to calculate a chi-square
	// with respect to the current interation as a progress monitor. At present this
	// is not very successful. 
	int usegold = 1;   
	thrustHvec<float> gold(vol_size);    // gold standard answer from simuation
	thrustDvec<float> dev_gold(vol_size);   // a chisqd will be calculated if the
	thrustDvec<float> dev_chisd(vol_size);  // file gold.raw is successfully read	
	if( cx::read_raw("gold.raw",gold.data(),vol_size,0) ) usegold = 0;  
	if (usegold) dev_gold = gold; 
	
	thrustHvec<float>     norm(norm_size); // voxel normaliztions depends on both ring and z values
	thrustDvec<float> dev_norm(norm_size);


	cx::timer ntim;
	ntim.start();

	// due to z-sliding of sm elements normaisation of sm elements requires care
	// the required factors are calculated here every run but could be read from
	// the saved file to save time.
	normalise_sm(sm,norm,systab,dzcut,valcut);
	cx::write_raw("smnorm.raw",norm.data(),norm_size);
	dev_norm = norm;
	ntim.add();
	printf("Host normalize call %.3f ms\n",ntim.time());

	double tot_activity = 0.0;
	for(uint k = 0; k < lor_size; k++) tot_activity += meas[k];

	//float mean_activity = tot_activity / vol_size;
	//for (uint k = 0; k < vol_size; k++) vol[k] = mean_activity;

	// new initialisation accounting for voxel volumes (makes little difference)
	float roi_volume = cx::pi<float>*roiRadius*roiRadius;
	float act_density = tot_activity/roi_volume;

	float r1 = 0.0f;
	float r2 = voxSize;
	for(int r=0;r<radNum;r++){
		float dr2 = r2*r2-r1*r1;
		float voxvol = cx::pi<float>*dr2/cryNum;
		for(uint k=0;k<zphi_size;k++) vol[r*zphi_size+k] = act_density*voxvol;
		r1 = r2;
		r2 += voxSize;
	}

	dev_vol = vol;
	printf("total activity %.0f, activity density %.0f\n",tot_activity,act_density);
	//cx::write_raw("reco_start_vol.raw",vol.data(),vol_size); // debug

	cx::timer tim1;
	cx::timer tim2;
	cx::timer tim3;
	cx::timer tim4;
	cx::timer all;

	all.reset();
	if(usehost)for(int iter = 0;iter< niter;iter++){
		if(iter>0){
			std::fill(FP.begin(),FP.end(),0);
			std::fill(BP.begin(),BP.end(),0);
		}
		tim1.reset();
		for(int r = rmin; r < rmax; r++) {
			forward_project_host(sm.data(),systab[r].start,systab[r].end,meas.data(),vol.data(),r,FP.data(),dzcut,valcut);
		}
		tim1.add();
		tim2.reset();
		for(int r = rmin; r < rmax; r++) {
			backward_project_host(sm.data(),systab[r].start,systab[r].end,meas.data(),r,FP.data(),BP.data(),dzcut,valcut);
		}
		tim2.add();
		tim3.reset();
		rescale_host(vol.data(),BP.data(),norm.data());
		tim3.add();

		all.add();
		printf("host iteration %3d times fwd %.3f bwd %.3f rsc %.3f all %.3f ms\n",iter+1,tim1.time(),tim2.time(),tim3.time(),all.time());
	}

	else for(int iter = 0;iter< niter;iter++){
		if(iter>0){
			clear_vector<float><<<blscale,thscale>>>(dev_FP.data().get(),lor_size);
			clear_vector<float><<<blscale,thscale>>>(dev_BP.data().get(),vol_size);
		}
		tim1.start();

		for(int r = rmin; r < rmax; r++) {
			forward_project<<<blocks,threads>>>(dev_sm.data().get(),systab[r].start,systab[r].end,dev_vol.data().get(),r,dev_FP.data().get(),dzcut,valcut);
		}
                cx::cudaOK(hipDeviceSynchronize());
                tim1.add();
		tim2.start();

		for(int r = rmin; r < rmax; r++) {
			backward_project<<<blocks,threads>>>(dev_sm.data().get(),systab[r].start,systab[r].end,dev_meas.data().get(),r,dev_FP.data().get(),dev_BP.data().get(),dzcut,valcut);
		}
                cx::cudaOK(hipDeviceSynchronize());
                tim2.add();
		tim3.start();
		rescale<<<blscale,thscale>>>(dev_vol.data().get(),dev_BP.data().get(),dev_norm.data().get());
                cx::cudaOK(hipDeviceSynchronize());
                tim3.add();
		// save  progress
		if ((iter+1)%snapsave == 0 && (iter+1) < niter) {
			vol = dev_vol;
			//sprintf(name, "%s%3.3d.raw", argv[2], iter+1);
			//cx::write_raw(name, vol.data(), vol_size);
			sprintf(name, "%s_cart%3.3d.raw", argv[2], iter+1);
			pol_save(name, vol.data());
		}
                cx::cudaOK(hipDeviceSynchronize());

                tim4.start();
		float xhisd  = 0.0f;
		if(usegold){
			calc_chisd<<<blscale,thscale>>>(dev_vol.data().get(),dev_gold.data().get(),dev_chisd.data().get());
                        cx::cudaOK(hipDeviceSynchronize());
                        xhisd = thrust::reduce(dev_chisd.begin(),dev_chisd.end());
		}
		tim4.add();

		all.add();
		printf("iteration %3d chi %5e times fwd %.3f bwd %.3f rsc %.3f chi %.3f all %.3f ms\n",iter+1,xhisd,tim1.time(),tim2.time(),tim3.time(),tim4.time(),all.time());
		int iout = iter+1;
		//if(iout<=5 || (iout<=10 && iout%2==0) || (iout<=50 && iout%10==0) || iout%50==0 || iout==niter ){  // long runs
		if(iout==niter){		
			vol = dev_vol;
			//sprintf(name,"%s_mlem%3.3d.raw",argv[2],iout);
			//cx::write_raw(name,vol.data(),vol_size);
			sprintf(name,"%s_cart%3.3d.raw",argv[2],iout);
			pol_save(name, vol.data());
		}
	}

	all.add();
	printf("All time %.3f ms\n",all.time());

	// these for debug 
	//if(!usehost)vol = dev_vol;
	//sprintf(name,"%s_%d_mlem.raw",argv[2],niter);
	//cx::write_raw(name, vol.data(), vol_size);

	//sprintf(name,"FPbug%3.3d.raw",niter);
	//FP = dev_FP;
	//cx::write_raw(name, FP.data(), lor_size);

	//BP = dev_BP;
	//sprintf(name,"BPbug%3.3d.raw",niter);
	//cx::write_raw(name, BP.data(), vol_size);

	return 0;
}
